
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>



#define CUDACHECK(err) { cuda_check((err), __FILE__, __LINE__); }
inline void cuda_check(hipError_t error_code, const char *file, int line)
{
    if (error_code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error %d: %s. In '%s' on line %d\n", error_code, hipGetErrorString(error_code), file, line);
        exit(error_code);
    }
}



void check_row_sums(unsigned int * row_sums, size_t n_rows, size_t n_cols)
{
    size_t error_count = 0;
    for(size_t r = 0; r < n_rows; r++)
    {
        int observed = row_sums[r];
        int correct = n_cols * (n_cols - 1) / 2 + 2 * r * n_cols;
        if(observed != correct)
        {
            if(error_count < 5)
                printf("Incorrect sum in row %ld. Correct is %d, but observed %d\n", r, correct, observed);
            error_count++;
        }
    }

    if(error_count == 0)
        printf("Row sum seems OK\n");
    else
        printf("Total errors: %d\n", error_count);
}

void check_col_sums(unsigned int * col_sums, size_t n_rows, size_t n_cols)
{
    size_t error_count = 0;
    for(size_t c = 0; c < n_cols; c++)
    {
        int observed = col_sums[c];
        int correct = 2 * (n_rows * (n_rows - 1) / 2) + c * n_rows;
        if(observed != correct)
        {
            if(error_count < 5)
                printf("Incorrect sum in col %ld. Correct is %d, but observed %d\n", c, correct, observed);
            error_count++;
        }
    }

    if(error_count == 0)
        printf("Col sum seems OK\n");
    else
        printf("Total errors: %d\n", error_count);
}







__global__ void matrix_init(unsigned int * matrix, size_t n_rows, size_t n_cols, size_t ld)
{
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;

    if(col < n_cols && row < n_rows)
        matrix[row * ld + col] = 2 * row + col;
}


// TODO kernely
__global__ 
void add_matrix_row(unsigned int * matrix, unsigned int * rows, size_t size, int ld, int pitch){
    size_t row = (blockIdx.x * blockDim.x + threadIdx.x); 
    
    if(row > size)
        return;
    
    size_t sum = 0; 
    for(size_t i = 0; i < size; ++i) {
        sum += matrix[row*ld+i];
    }
    
    rows[row] = sum;
}


__global__ 
void add_matrix_col(unsigned int * matrix, unsigned int * cols, size_t size, int ld, int pitch){
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(col > size)
        return;
    
    size_t sum = 0; 
    for(size_t i = 0; i < size; ++i) {
        sum += matrix[col + i*ld];
    }
    
    cols[col] = sum;
}




int main()
{
    size_t size = 98765;

    unsigned int * d_matrix;
    size_t d_pitch;
    CUDACHECK(hipMallocPitch(&d_matrix, &d_pitch, size * sizeof(unsigned int), size));
    size_t d_ld = d_pitch / sizeof(int);
    // d_ld = size;

    unsigned int * d_row_sums;
    CUDACHECK(hipMalloc(&d_row_sums, size * sizeof(unsigned int)));

    unsigned int * d_col_sums;
    CUDACHECK(hipMalloc(&d_col_sums, size * sizeof(unsigned int)));

    unsigned int * h_row_sums;
    unsigned int * h_col_sums;
    CUDACHECK(hipHostMalloc(&h_row_sums, size * sizeof(unsigned int), hipHostMallocDefault));
    CUDACHECK(hipHostMalloc(&h_col_sums, size * sizeof(unsigned int), hipHostMallocDefault));
    
    hipEvent_t start_init, end_init, start_rows, end_rows, start_cols, end_cols;
    CUDACHECK(hipEventCreate(&start_init));
    CUDACHECK(hipEventCreate(&end_init));
    CUDACHECK(hipEventCreate(&start_rows));
    CUDACHECK(hipEventCreate(&end_rows));
    CUDACHECK(hipEventCreate(&start_cols));
    CUDACHECK(hipEventCreate(&end_cols));

    dim3 tpb, bpg;
    


    tpb = dim3(32, 32);
    bpg = dim3((size - 1) / tpb.x + 1, (size - 1) / tpb.y + 1);
    CUDACHECK(hipEventRecord(start_init));
    matrix_init<<< bpg, tpb >>>(d_matrix, size, size, d_ld);
    CUDACHECK(hipEventRecord(end_init));

    tpb = 512;
    bpg = (size - 1) / tpb.x + 1;
    hipEventRecord(start_rows);
    add_matrix_row<<<bpg, tpb>>>(d_matrix, d_row_sums, size, d_ld, d_pitch);
    hipEventRecord(end_rows);

    tpb = 512;
    bpg = (size - 1) / tpb.x + 1;
    hipEventRecord(start_cols);
    add_matrix_col<<<bpg, tpb>>>(d_matrix, d_col_sums, size, d_ld, d_pitch);
    hipEventRecord(end_cols);


    CUDACHECK(hipMemcpy(h_row_sums, d_row_sums, size * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDACHECK(hipMemcpy(h_col_sums, d_col_sums, size * sizeof(unsigned int), hipMemcpyDeviceToHost));

    check_row_sums(h_row_sums, size, size);
    check_col_sums(h_col_sums, size, size);



    float time_init, time_rows, time_cols;
    
    
    // TODO: find the time
    hipEventElapsedTime(&time_init, start_init, end_init);
    hipEventElapsedTime(&time_rows, start_rows, end_rows);
    hipEventElapsedTime(&time_cols, start_cols, end_cols);
    printf("\n");
    printf("Matrix init time:              %7.3f ms\n", time_init);
    printf("Summation time in each row:    %7.3f ms\n", time_rows);
    printf("Summation time in each column: %7.3f ms\n", time_cols);
    printf("Using coalesced memory accesses was %5.2f times faster\n", time_rows / time_cols);



    CUDACHECK(hipEventDestroy(start_init));
    CUDACHECK(hipEventDestroy(end_init));
    CUDACHECK(hipEventDestroy(start_rows));
    CUDACHECK(hipEventDestroy(end_rows));
    CUDACHECK(hipEventDestroy(start_cols));
    CUDACHECK(hipEventDestroy(end_cols));

    CUDACHECK(hipFree(d_matrix));
    CUDACHECK(hipFree(d_row_sums));
    CUDACHECK(hipFree(d_col_sums));
    CUDACHECK(hipHostFree(h_row_sums));
    CUDACHECK(hipHostFree(h_col_sums));

    return 0;
}
