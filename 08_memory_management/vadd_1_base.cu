
#include <hip/hip_runtime.h>
#include <cstdio>
#include <algorithm>



#define CUDACHECK(err) { cuda_check((err), __FILE__, __LINE__); }
inline void cuda_check(hipError_t error_code, const char *file, int line)
{
    if (error_code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error %d: %s. '%s' line %d\n", error_code, hipGetErrorString(error_code), file, line);
        exit(error_code);
    }
}



void print_vector(float * data, int count, const char * label)
{
    int print_max = 20;
    int print_count = std::min(count, print_max);

    printf("%s:\n", label);
    for(int i = 0; i < print_count; i++)
        printf("%7.3f ", data[i]);
    printf("\n");
}

void check_result(float * a, float * b, float * c, int count)
{
    int errorCount = 0;
    for(int i = 0; i < count; i++)
    {
        if(c[i] != a[i] + b[i])
        {
            errorCount++;
            if(errorCount <= 5)
            {
                printf("Error on index %d: correct is %f, but result is %f\n", i, a[i] + b[i], c[i]);
            }
        }
    }
    if(errorCount == 0)
    {
        printf("The result is CORRECT!\n");
    }
}


__global__
void vecAdd(float* a, float *b, float *c, int start, int end)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x + start;
 
    // Boundary
    if (id < end)
        c[id] = a[id] + b[id];
}


int main()
{
    int count = 1234567;
    size_t size = sizeof(float)*count;

    float* a;
    hipMallocManaged(&a, size);
    float* b;
    hipMallocManaged(&b, size);
    float* c;
    hipMallocManaged(&c, size);

    for(int i = 0; i < count; i++)
        a[i] = i;
    for(int i = 0; i < count; i++)
        b[i] = 10 * i;
    print_vector(a, count, "Input A");
    print_vector(b, count, "Input B");

    int devs = 0;
    hipGetDeviceCount(&devs);
    hipDeviceSynchronize();

    int stride = (count / devs);
    int start = 0;
    int end = stride;
    for(int i = 0; i < devs; ++i){
        hipSetDevice(i);

        if(i == devs-1)
            end = count;
        vecAdd<<<(end-start-1)/32+1,32>>>(a, b, c, start, end);

        printf("s: %d, e:%d\n", start, end);
        start = end;
        end += stride; 
    }

    for(int i = 0; i < devs; ++i){
        hipSetDevice(i);
        hipDeviceSynchronize();
    }

    print_vector(c, count, "Output C");
    check_result(a, b, c, count);

    CUDACHECK(hipFree(a));
    CUDACHECK(hipFree(b));
    CUDACHECK(hipFree(c));

    return 0;
}
