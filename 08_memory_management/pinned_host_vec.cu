#include "hip/hip_runtime.h"
#include <cstdio>
#include <algorithm>
#include "timer.h"


#define CUDACHECK(err) { cuda_check((err), __FILE__, __LINE__); }
inline void cuda_check(hipError_t error_code, const char *file, int line)
{
    if (error_code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error %d: %s. '%s' line %d\n", error_code, hipGetErrorString(error_code), file, line);
        exit(error_code);
    }
}

void print_vector(float * data, int vector_length, const char * label)
{
    int print_max = 20;
    int print_vector_length = std::min(vector_length, print_max);

    printf("%s:\n", label);
    for(int i = 0; i < print_vector_length; i++)
        printf("%7.3f ", data[i]);
    printf("\n");
}

void check_result(float *a, float *b, float *c, int vector_length)
{
    int errorvector_length = 0;
    for(int i = 0; i < vector_length; i++)
    {
        if(c[i] != a[i] + b[i])
        {
            errorvector_length++;
            if(errorvector_length <= 5)
            {
                printf("Error on index %d: correct is %f, but result is %f\n", i, a[i] + b[i], c[i]);
            }
        }
    }
    if(errorvector_length == 0)
    {
        printf("The result is CORRECT!\n");
    }
}


__global__
void vecAdd(float *a, float *b, float *c, int start, int end)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x + start;

    // Boundary
    if (id < end)
        c[id] = a[id] + b[id];
}


int main()
{
    int vector_length = 1234567;
    size_t size = sizeof(float)*vector_length;

    float* a;
    hipHostMalloc(&a, size);
    float* b;
    hipHostMalloc(&b, size);
    float* c;
    hipHostMalloc(&c, size);

    float *d_a;
    hipHostMalloc(&a, size);
    float* b;
    hipHostMalloc(&b, size);
    float* c;
    hipHostMalloc(&c, size);

    for(int i = 0; i < vector_length; i++)
        a[i] = i;
    for(int i = 0; i < vector_length; i++)
        b[i] = 10 * i;
    print_vector(a, vector_length, "Input A");
    print_vector(b, vector_length, "Input B");

    TIMER_BEGIN("managed test begin");

    hipDeviceSynchronize();
        vecAdd<<<(end-start-1)/32+1, 32>>>(a, b, c, start, end);

     TIMER_END("managed test end");


    print_vector(c, vector_length, "Output C");
    check_result(a, b, c, vector_length);

    CUDACHECK(hipFree(a));
    CUDACHECK(hipFree(b));
    CUDACHECK(hipFree(c));

    return 0;
}
