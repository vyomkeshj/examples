#include "hip/hip_runtime.h"
#include <cstdio>
#include <algorithm>



#define CUDACHECK(err) { cuda_check((err), __FILE__, __LINE__); }
inline void cuda_check(hipError_t error_code, const char *file, int line)
{
    if (error_code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error %d: %s. '%s' line %d\n", error_code, hipGetErrorString(error_code), file, line);
        exit(error_code);
    }
}



void print_matrix(float * data, int n_rows, int n_cols, const char * label)
{
    int print_max_size = 8;
    int print_rows = std::min(n_rows, print_max_size);
    int print_cols = std::min(n_cols, print_max_size);

    printf("%s:\n", label);
    for(int row = 0; row < print_rows; row++)
    {
        for(int col = 0; col < print_cols; col++)
        {
            float value = data[row * n_cols + col];
            printf("%7.1f ", value);
        }
        printf("\n");
    }
}

void init_matrix(float * matrix, int n_rows, int n_cols)
{
    for(int row = 0; row < n_rows; row++)
    {
        for(int col = 0; col < n_cols; col++)
        {
            matrix[row * n_cols + col] = 10 * row + col;
        }
    }
}

void check_result(float * matrix, int n_rows, int n_cols, float scalar)
{
    int errorCount = 0;
    for(int row = 0; row < n_rows; row++)
    {
        for(int col = 0; col < n_cols; col++)
        {
            float result = matrix[row * n_cols + col];
            float correct = scalar * (10 * row + col);
            if(result != correct)
            {
                errorCount++;
                if(errorCount <= 5)
                    printf("Incorrect result row %d col %d: correct is %7.1f, but result is %7.1f\n", row, col, correct, result);
            }
        }
    }
    if(errorCount == 0)
    {
        printf("The result is CORRECT!\n");
    }
}



// TODO: matrix scale kernel
__global__ void scale_matrix()


int main()
{
    float scalar = 10.0f;
    int n_rows = 642;
    int n_cols = 531;
    int total_elements = n_rows * n_cols;

    float * h_matrix;
    // TODO: allocate
    init_matrix(h_matrix, n_rows, n_cols);
    print_matrix(h_matrix, n_rows, n_cols, "Input");

    float * d_matrix;
    
    // TODO ...

    print_matrix(h_matrix, n_rows, n_cols, "Output");

    check_result(h_matrix, n_rows, n_cols, scalar);

    // TODO

    return 0;
}
